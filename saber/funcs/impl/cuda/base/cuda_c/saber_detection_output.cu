#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_detection_output.h"
#include "saber/funcs/impl/detection_helper.h"
namespace anakin{

namespace saber{
template <typename dtype>
__global__ void permute_data_kernel(const int nthreads,
                                  const dtype* data, const int num_classes, const int num_data,
                                  const int num_dim, dtype* new_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int i = index % num_dim;
        const int c = (index / num_dim) % num_classes;
        const int d = (index / num_dim / num_classes) % num_data;
        const int n = index / num_dim / num_classes / num_data;
        const int new_index = ((n * num_classes + c) * num_data + d) * num_dim + i;
        new_data[new_index] = data[index];
    }
}

template <typename dtype>
void permute_data(const int nthreads,
                    const dtype* data, const int num_classes, const int num_data,
                    const int num_dim, dtype* new_data, hipStream_t stream) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    permute_data_kernel<dtype><<<CUDA_GET_BLOCKS(nthreads),
            CUDA_NUM_THREADS, 0, stream>>>(nthreads, data, num_classes, num_data, num_dim, new_data);
}

template <DataType OpDtype>
SaberStatus SaberDetectionOutput<NV, OpDtype>::dispatch(const std::vector<Tensor<NV> *>& inputs,
    std::vector<Tensor<NV> *>& outputs,
    DetectionOutputParam<NV>& param) {

    hipStream_t stream = this->_ctx->get_compute_stream();

    Tensor<NV>* t_loc = inputs[0];
    Tensor<NV>* t_conf = inputs[1];
    Tensor<NV>* t_prior = inputs[2];

    const dtype* loc_data = static_cast<const dtype*>(t_loc->data());
    const dtype* prior_data = static_cast<const dtype*>(t_prior->data());
    const int num = t_loc->num();

    // Decode predictions.
    dtype* bbox_data = static_cast<dtype*>(_bbox_preds.mutable_data());
    const int loc_count = _bbox_preds.valid_size();
    decode_bboxes<dtype>(loc_count, loc_data, prior_data, param.type, \
        param.variance_encode_in_target, _num_priors, param.share_location, \
        _num_loc_classes, param.background_id, bbox_data, stream);
    // Retrieve all decoded location predictions.
    if (!param.share_location) {
        dtype * bbox_permute_data = static_cast<dtype*>(_bbox_permute.mutable_data());
        permute_data<dtype>(loc_count, bbox_data, _num_loc_classes, _num_priors,
                              4, bbox_permute_data, stream);
    }
    // Retrieve all confidences.
    dtype* conf_permute_data = static_cast<dtype*>(_conf_permute.mutable_data());
    permute_data<dtype>(t_conf->valid_size(), static_cast<dtype*>(t_conf->data()), \
         this->_num_classes, _num_priors, 1, conf_permute_data, stream);

    CUDA_CHECK(hipMemcpyAsync(_bbox_cpu_data, static_cast<dtype*>(_bbox_preds.data()), \
                _bbox_preds.valid_size() * sizeof(dtype), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(_conf_cpu_data, static_cast<dtype*>(_conf_permute.data()), \
                _conf_permute.valid_size() * sizeof(dtype), hipMemcpyDeviceToHost, stream));
    hipStreamSynchronize(stream);

    std::vector<dtype> result;

    nms_detect(_bbox_cpu_data, _conf_cpu_data, result, num, this->_num_classes, _num_priors, param.background_id, \
        param.keep_top_k, param.nms_top_k, param.conf_thresh, param.nms_thresh, param.nms_eta, param.share_location);

    if(result.size() == 0) {
        result.resize(7);
        for (int i = 0; i < 7; ++i) {
            result[i] = (dtype)-1;
        }
        outputs[0]->reshape(Shape({1, 1, 1, 7}));
    } else {
        outputs[0]->reshape(Shape({1, 1, static_cast<int>(result.size() / 7), 7}));
    }

    CUDA_CHECK(hipMemcpyAsync(outputs[0]->mutable_data(), result.data(), \
                result.size() * sizeof(dtype), hipMemcpyHostToDevice, stream));

    return SaberSuccess;
}

//template class SaberDetectionOutput<AK_FLOAT, NCHW>;
} //namespace anakin

} //namespace anakin
